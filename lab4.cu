#include "hip/hip_runtime.h"
#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define MASK_WIDTH 3
#define BLOCK_SIZE 6
#define TILE_WIDTH 4
//@@ Define constant memory for device kernel here
__constant__ float Mask[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here

  __shared__ float N_ds[BLOCK_SIZE][BLOCK_SIZE][BLOCK_SIZE];
  int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z;
  int bx = blockIdx.x, by = blockIdx.y, bz = blockIdx.z;
  
  int z_o = bz * TILE_WIDTH + tz;
  int y_o = by * TILE_WIDTH + ty;
  int x_o = bx * TILE_WIDTH + tx;
  
  int z_i = z_o -(MASK_WIDTH/2);
  int y_i = y_o -(MASK_WIDTH/2);
  int x_i = x_o -(MASK_WIDTH/2);
  

  if((z_i >=0 && z_i < z_size)&&(y_i >=0 && y_i < y_size)&&(x_i >=0 && x_i < x_size)){
    N_ds[tz][ty][tx] = input[z_i * y_size * x_size + y_i * x_size + x_i];
  }
  else{
    N_ds[tz][ty][tx] = 0.0f;
  }

  __syncthreads();
  
  float pValue = 0.0f;
  
  if(tz < TILE_WIDTH && ty < TILE_WIDTH && tx < TILE_WIDTH){
    for(int i = 0; i < MASK_WIDTH; i++){
      for(int j = 0; j < MASK_WIDTH; j++){
        for(int k = 0; k < MASK_WIDTH; k++){
          if(z_o == 0 && y_o == 0 && x_o == 28){
              printf("Detected in loop");
          }
          pValue += Mask[i][j][k] * N_ds[i+tz][j+ty][k+tx];
        }
      }
    }
    
    if((z_o >=0 && z_o < z_size)&&(y_o >=0 && y_o < y_size)&&(x_o >=0 && x_o < x_size)){
      output[z_o * y_size * x_size + y_o * x_size + x_o] = pValue;
    }
    
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);
  
  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void**)&deviceInput, z_size*y_size*x_size*sizeof(float));
  hipMalloc((void**)&deviceOutput, z_size*y_size*x_size*sizeof(float));
  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInput, hostInput+3,z_size*y_size*x_size*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(Mask), hostKernel, kernelLength*sizeof(float));

  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
  dim3 DimGrid(ceil((x_size*1.0)/TILE_WIDTH),ceil((y_size*1.0)/TILE_WIDTH),ceil((z_size*1.0)/TILE_WIDTH));
  //@@ Launch the GPU kernel here
  conv3d<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy(hostOutput+3, deviceOutput, z_size*y_size*x_size*sizeof(float),hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");
  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);
  
  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}
